# include <stdio.h>
# include <stdlib.h>
# include <string.h>
# include <hip/hip_runtime.h>

// # include <time.h> //提供了处理日期和时间的标准函数和数据结构
# include <sys/time.h> // 通常用于获取更加精确的时间信息

void sumArrayOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; i++)
        C[i] = A[i] + B[i];
}

__global__ void sumArrayOnDevice(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);
}

int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const int N = 1<<24;
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *gpu_C = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    memset(h_C, 0, size);
    memset(gpu_C, 0, size);

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    double iStart, iElaps;
    iStart=cpuSecond();

    dim3 block(256);
    dim3 grid((N - 1) / block.x + 1);
    sumArrayOnDevice<<<grid,block>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;

    hipMemcpy(gpu_C, d_C, size, hipMemcpyDeviceToHost);

    double iStart_cpu,iElaps_cpu;
    iStart_cpu = cpuSecond();
    sumArrayOnHost(h_A, h_B, h_C, N);
    iElaps_cpu = cpuSecond() - iStart_cpu;

    for (int i = 0; i < N; i++) {
        if (gpu_C[i] != h_C[i]) {
            printf("Error: gpu_C[%d] = %f != %f = h_C[%d]\n", i, gpu_C[i], h_C[i], i);
            exit(EXIT_FAILURE);
        }
    }

    printf("cup time cost: %f\ngpu time cost: %f\n", iElaps_cpu, iElaps);

    free(h_A);
    free(h_B);
    free(h_C);
    free(gpu_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
 
    return 0;
}