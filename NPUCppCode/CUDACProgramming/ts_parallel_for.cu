# include <stdio.h>
# include <hip/hip_runtime.h>

void sumArrayOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; i++)
        C[i] = A[i] + B[i];
}

__global__ void sumArrayOnDevice(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const int N = 1024;
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *gpu_C = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    memset(h_C, 0, size);
    memset(gpu_C, 0, size);

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    sumArrayOnDevice<<<1,N>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    hipMemcpy(gpu_C, d_C, size, hipMemcpyDeviceToHost);

    sumArrayOnHost(h_A, h_B, h_C, N);

    for (int i = 0; i < N; i++) {
        if (gpu_C[i] != h_C[i]) {
            printf("Error: gpu_C[%d] = %f != %f = h_C[%d]\n", i, gpu_C[i], h_C[i], i);
            exit(EXIT_FAILURE);
        }
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(gpu_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
        
}
