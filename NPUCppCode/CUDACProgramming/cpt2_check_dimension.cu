// Code 2-2
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void CheckIndex()
{
    // 核函数中，每个线程都输出自己的线程索引，块索引，块维度，网格维度
    printf("threadIdx: (%d, %d, %d) \t\
            blockIdx: (%d, %d, %d) \t\
            blockDim: (%d, %d, %d) \t\
            gridDim: (%d, %d, %d)\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    int n_element = 6;

    dim3 block = 3;
    dim3 grid = ((n_element + block.x - 1) / block.x);

    // 主机端，检查块和网格维度
    printf("grid.x =  %d, grid.y =  %d, grid.z =  %d\n", grid.x, grid.y, grid.z);
    printf("block.x = %d, block.y = %d, block.z = %d\n", block.x, block.y, block.z);

    CheckIndex<<<grid, block>>>();
    hipDeviceReset();

    return 0;
}