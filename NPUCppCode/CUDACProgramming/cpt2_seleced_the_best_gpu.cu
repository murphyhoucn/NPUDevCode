// Code 2-9
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int num_device = 0;
    hipGetDeviceCount(&num_device);

    if (num_device > 1)
    {
        int max_multiprocessors = 0, max_device = 0;
        for (int device = 0; device < num_device; ++device)
        {
            hipDeviceProp_t device_prop;
            hipGetDeviceProperties(&device_prop, device);

            if (max_multiprocessors < device_prop.multiProcessorCount)
            {
                max_multiprocessors = device_prop.multiProcessorCount;
                max_device = device;
            }
        }

        hipSetDevice(max_device);
    }
    else
    {
        hipSetDevice(0);
    }

    return 0;
}