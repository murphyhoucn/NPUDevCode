// Code 2-6
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                                                               \
    {                                                                                                             \
        const hipError_t error = call;                                                                           \
        if (error != hipSuccess)                                                                                 \
        {                                                                                                         \
            printf("Error %s: %d, code: %d, reason: %s\n", __FILE__, __LINE__, error, hipGetErrorString(error)); \
            exit(-10 * error);                                                                                    \
        }                                                                                                         \
    }

void InitialInt(int *ip, int size)
{
    for (int i = 0; i < size; ++i)
        ip[i] = i;
}

void PrintMatrix(int *c, const int knx, const int kny)
{
    int *ic = c;
    printf("\nMatrix: (%d, %d)\n", knx, kny);

    for (int iy = 0; iy < kny; ++iy)
    {
        for (int ix = 0; ix < knx; ++ix)
        {
            printf("%3d", ic[ix]);
        }
        ic += knx;
        printf("\n");
    }
    printf("\n");
}

__global__ void PrintThreadIndex(int *a, const int knx, const int kny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * knx + ix;

    printf("thread_id:(%d, %d) block_id:(%d, %d) coordinate:(%d, %d) global index:%d ival:%d\n",
           threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, a[idx]);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // get device infomation
    int dev = 0;
    hipDeviceProp_t device_prop;
    CHECK(hipGetDeviceProperties(&device_prop, dev));
    printf("Using device %d:%s\n", dev, device_prop.name);
    CHECK(hipSetDevice(dev));

    // set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int n_bytes = nxy * sizeof(float);

    // malloc host memory
    int *h_a;
    h_a = (int *)malloc(n_bytes);

    // initialize host matrix with integer
    InitialInt(h_a, nxy);
    PrintMatrix(h_a, nx, ny);

    // malloc device memory
    int *d_matrix_a;
    hipMalloc((void **)&d_matrix_a, n_bytes);

    // transfer data from host to device
    hipMemcpy(d_matrix_a, h_a, n_bytes, hipMemcpyHostToDevice);

    // set up execution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invoke the kernel
    PrintThreadIndex<<<grid, block>>>(d_matrix_a, nx, ny);
    hipDeviceSynchronize();

    // free host and device memory
    free(h_a);
    hipFree(d_matrix_a);

    // reset device
    hipDeviceReset();

    return 0;
}