// Code 2-1

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

void SumArrayOnHost(float *array_a, float *array_b, float *array_c, const int kN)
{
    for (int idx = 0; idx < kN; ++idx)
    {
        array_c[idx] = array_a[idx] + array_b[idx];
    }
}

void InitialArray(float *ip, int size)
{
    time_t t;
    srand((unsigned int)time(&t));

    for (int i = 0; i < size; ++i)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

int main()
{
    int n_element = 1024;
    size_t n_bytes = n_element * sizeof(float);

    float *h_array_a, *h_array_b, *h_array_c;
    h_array_a = (float *)malloc(n_bytes);
    h_array_b = (float *)malloc(n_bytes);
    h_array_c = (float *)malloc(n_bytes);

    InitialArray(h_array_a, n_element);
    InitialArray(h_array_b, n_element);

    SumArrayOnHost(h_array_a, h_array_b, h_array_c, n_element);

    free(h_array_a);
    free(h_array_b);
    free(h_array_c);

    return 0;
}