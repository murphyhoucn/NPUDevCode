// Code 1-1

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    int thread_index_x = threadIdx.x;

    printf("Hello world from GPU! thread: %d\n", thread_index_x);
}

int main()
{
    printf("Hello world from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    // cudaDeviceSynchronize();

    return 0;
}