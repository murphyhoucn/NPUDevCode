#include "hip/hip_runtime.h"
// 习题 1
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// '/'的意思是换行符，一行代码写成了多行
#define CHECK(call)                                                             \
    {                                                                           \
        const hipError_t error = call;                                         \
        if (error != hipSuccess)                                               \
        {                                                                       \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
            printf("cude: %d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                            \
        }                                                                       \
    }

void CheckResult(float *host_ref, float *gpu_ref, const int kN)
{
    double epsilon = 1.0e-8;
    bool match = 1;

    for (int i = 0; i < kN; ++i)
    {
        if (abs(host_ref[i] - gpu_ref[i]) > epsilon)
        {
            match = 0;
            printf("Array do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", host_ref[i], gpu_ref[i], i);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
}

void InitData(float *ip, int size)
{
    // generate different seed for random number.
    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; ++i)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

double CpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.0e-6);
}

void SumArrayOnHost(float *a, float *b, float *c, const int kN)
{
    for (int idx = 0; idx < kN; ++idx)
    {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void SumArrayOnDevice(float *a, float *b, float *c, const int kN)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < kN)
        c[i] = a[i] + b[i];
}

int main()
{
    // set device
    int dev = 0;
    hipDeviceProp_t device_prop;
    CHECK(hipGetDeviceProperties(&device_prop, dev));
    printf("Using device %d: %s\n", dev, device_prop.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vector
    int n_element = 1 << 24;
    printf("Vector size %d\n", n_element);

    // malloc host memory
    size_t n_bytes = n_element * sizeof(float);

    float *h_a, *h_b, *host_ref, *gpu_ref;
    h_a = (float *)malloc(n_bytes);
    h_b = (float *)malloc(n_bytes);
    host_ref = (float *)malloc(n_bytes);
    gpu_ref = (float *)malloc(n_bytes);

    double i_strat, i_elaps;

    // initialize data at host side
    i_strat = CpuSecond();
    InitData(h_a, n_element);
    InitData(h_b, n_element);
    i_elaps = CpuSecond() - i_strat;

    memset(host_ref, 0, n_bytes);
    memset(gpu_ref, 0, n_bytes);

    // malloc device global memoty
    float *d_a, *d_b, *d_c;
    hipMalloc((float **)&d_a, n_bytes);
    hipMalloc((float **)&d_b, n_bytes);
    hipMalloc((float **)&d_c, n_bytes);

    // transfer data from host to device
    hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    int i_lens = 1024;
    dim3 block(i_lens);
    dim3 grid((n_element + block.x - 1) / block.x);

    i_strat = CpuSecond();
    SumArrayOnDevice<<<grid, block>>>(d_a, d_b, d_c, n_element);
    i_elaps = CpuSecond() - i_strat;
    printf("Execution configuration<<<%d, %d>>> Time eplapsed %f sec\n", grid.x, block.x, i_elaps);

    // copy kernel result back to host side
    hipMemcpy(gpu_ref, d_c, n_bytes, hipMemcpyDeviceToHost);

    // add vector at host side for result checks
    i_strat = CpuSecond();
    SumArrayOnHost(h_a, h_b, host_ref, n_element);
    i_elaps = CpuSecond() - i_strat;

    // check device results
    CheckResult(host_ref, gpu_ref, n_element);

    // free device global memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // free host memory
    free(h_a);
    free(h_b);
    free(host_ref);
    free(gpu_ref);

    // reset device
    hipDeviceReset();

    return 0;
}

/*
 *Using device 0: NVIDIA GeForce RTX 3080
 *Vector size 16777216
 *Execution configuration<<<16401, 1023>>> Time eplapsed 0.000504 sec
 *Arrays match.
 *
 *grid = (16401,1,1)
 *block = (1023,1,1)
 */

/*
 *Using device 0: NVIDIA GeForce RTX 3080
 *Vector size 16777216
 *Execution configuration<<<16384, 1024>>> Time eplapsed 0.000495 sec
 *Arrays match.
 *
 *grid = (16384,1,1)
 *block = (1024,1,1)
 */

/*
 *额。。。。不知道怎么分析，感觉运行速度差不多？？
 */