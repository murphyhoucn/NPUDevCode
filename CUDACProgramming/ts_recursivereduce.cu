#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common/common.h"

// CPU
int RecursiveReduce(int *data, const int size)
{
    if (size == 1)
        return data[0];

    const int stride = size / 2;

    if (size % 2 == 1)
    {
        for (int i = 0; i < stride; i++)
            data[i] += data[i + stride];
        data[0] += data[size - 1];
    }
    else
    {
        for (int i = 0; i < stride; i++)
            data[i] += data[i + stride];
    }

    return RecursiveReduce(data, stride);
}

// 相邻配对-warmup
__global__ void WarmUp(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    if (tid >= n)
        return;

    int *idata = g_idata + blockDim.x * blockIdx.x;

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride) == 0))
            idata[tid] += idata[tid + stride];

        __syncthreads(); // 同步
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

// 相邻配对
__global__ void ReduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    if (tid >= n)
        return;

    int *idata = g_idata + blockDim.x * blockIdx.x;

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride) == 0))
            idata[tid] += idata[tid + stride];

        __syncthreads(); // 同步
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void ReduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int *idata = g_idata + blockDim.x * blockIdx.x;

    if (idx > n)
        return;

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;
        if (index < blockDim.x)
            idata[index] += idata[index + stride];
        __syncthreads();
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void ReduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    int *idata = g_idata + blockDim.x * blockIdx.x;

    if (idx >= n)
        return;

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
            idata[tid] += idata[tid + stride];
        __syncthreads();
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char **argv)
{
    int devidx = 0;
    hipSetDevice(devidx);

    // bool bResult = false;

    // initialization

    int size = 1 << 24;
    printf("with array size %d  ", size);

    // execution configuration
    int blocksize = 1024;
    if (argc > 1)
    {
        blocksize = atoi(argv[1]);
    }
    dim3 block(blocksize, 1);
    dim3 grid((size - 1) / block.x + 1, 1);
    printf("grid %d block %d \n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *idata_host = (int *)malloc(bytes);
    int *odata_host = (int *)malloc(grid.x * sizeof(int));
    int *tmp = (int *)malloc(bytes);

    // initialize the array
    initialData_int(idata_host, size);

    memcpy(tmp, idata_host, bytes);
    double iStart, iElaps;
    int gpu_sum = 0;

    // device memory
    int *idata_dev = NULL;
    int *odata_dev = NULL;
    CHECK(hipMalloc((void **)&idata_dev, bytes));
    CHECK(hipMalloc((void **)&odata_dev, grid.x * sizeof(int)));

    // cpu reduction
    int cpu_sum = 0;
    iStart = cpuSecond();
    // cpu_sum = recursiveReduce(tmp, size);
    for (int i = 0; i < size; i++)
        cpu_sum += tmp[i];
    printf("cpu sum:%d \n", cpu_sum);
    iElaps = cpuSecond() - iStart;
    printf("cpu reduce                 elapsed %lf ms cpu_sum: %d\n", iElaps, cpu_sum);

    // kernel 1:reduceNeighbored

    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    WarmUp<<<grid, block>>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu warmup                 elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // kernel 1:reduceNeighbored

    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    ReduceNeighbored<<<grid, block>>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceNeighbored       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // kernel 2:reduceNeighboredLess

    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    ReduceNeighboredLess<<<grid, block>>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceNeighboredLess   elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // kernel 3:reduceInterleaved
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    ReduceInterleaved<<<grid, block>>>(idata_dev, odata_dev, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += odata_host[i];
    printf("gpu reduceInterleaved      elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);
    // free host memory

    free(idata_host);
    free(odata_host);
    CHECK(hipFree(idata_dev));
    CHECK(hipFree(odata_dev));

    // reset device
    hipDeviceReset();

    // check the results
    if (gpu_sum == cpu_sum)
    {
        printf("Test success!\n");
    }
    return EXIT_SUCCESS;
}
