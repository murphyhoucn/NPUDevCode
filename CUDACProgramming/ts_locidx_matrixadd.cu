#include "hip/hip_runtime.h"
# include <stdio.h>
# include <hip/hip_runtime.h>
# include <stdlib.h>
# include <string.h>

#include "common/common.h"

__global__ void printThreadIdx(float* A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    unsigned int idx = ix + nx * iy;
    printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d) global index %2d ival %2f\n",threadIdx.x,threadIdx.y, blockIdx.x,blockIdx.y, ix,iy, idx, A[idx]);

}

__global__ void sumMatrix(float* MA, float* MB, float* MC, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int ixy = ix + iy * nx;
    if (ix < nx && iy < ny)
        MC[ixy] = MA[ixy] + MB[ixy];
}

void sumMatrixCPU(float* MA, float* MB, float* MC, const int nx, const int ny)
{
    float* pa = MA;
    float* pb = MB;
    float* pc = MC;

    for(int j = 0; j < ny; j++)
    {
        for(int i = 0; i < nx; i++)
        {
            pc[i] = pa[i] + pb[i];
        }
        pa += nx;
        pb += nx;
        pc += nx;
    }
}

void checkResult(float * CPU, float * GPU, const int N)
{
    for (int i = 0; i < N; i++) {
        if (GPU[i] != CPU[i]) {
            printf("Error: gpu_C[%d] = %f != %f = h_C[%d]\n", i, GPU[i], CPU[i], i);
            exit(EXIT_FAILURE);
        }
    }
}

int main()
{
    int dev_idx = 0;
    hipSetDevice(dev_idx);

    srand((unsigned int)time(NULL));

    int nx = 1<<12, ny=1<<12;
    int nxy = nx * ny;
    size_t nBytes = nxy * sizeof(float);

    float* h_A = (float*)malloc(nBytes);
    float* h_B = (float*)malloc(nBytes);
    float* h_C = (float*)malloc(nBytes);
    float* h_C_from_gpu = (float*)malloc(nBytes);


    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = (float)rand() / RAND_MAX;
        h_B[i] = (float)rand() / RAND_MAX;
    }
    memset(h_C, 0, nBytes);
    memset(h_C_from_gpu, 0, nBytes);


    // device
    float* d_A = NULL;
    float* d_B = NULL;
    float* d_C = NULL;
    CHECK(hipMalloc((void**)&d_A, nBytes));
    CHECK(hipMalloc((void**)&d_B, nBytes));
    CHECK(hipMalloc((void**)&d_C, nBytes));
    
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // cpu section
    double iStart = Seconds();
    sumMatrixCPU(h_A, h_B, h_C, nx, ny);
    double iElaps = Seconds() - iStart;
    printf("CPU Execution Time elapsed %f sec\n",iElaps);

    // gpu section

    // 2d block and 2d grid
    int dimx=32;
    int dimy=32; 
    dim3 block1(dimx, dimy);
    dim3 grid1((nx - 1)/block1.x + 1 ,(ny - 1)/block1.y + 1);

    iStart=Seconds();
    sumMatrix<<<grid1,block1>>>(d_A, d_B, d_C, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps=Seconds()-iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n", grid1.x,grid1.y, block1.x,block1.y, iElaps);
    CHECK(hipMemcpy(h_C_from_gpu,d_C,nBytes,hipMemcpyDeviceToHost));
    checkResult(h_C, h_C_from_gpu, nxy);

    // 1d block and 1d grid
    dimx=32;
    dim3 block2(dimx);
    dim3 grid2((nxy-1)/block2.x+1);
    iStart=Seconds();
    sumMatrix<<<grid2,block2>>>(d_A,d_B,d_C,nx*ny,1);
    CHECK(hipDeviceSynchronize());
    iElaps=Seconds()-iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid2.x,grid2.y,block2.x,block2.y,iElaps);
    CHECK(hipMemcpy(h_C_from_gpu,d_C,nBytes,hipMemcpyDeviceToHost));
    checkResult(h_C,h_C_from_gpu,nxy);


    // 2d block and 1d grid
    dimx=32;
    dim3 block3(dimx);
    dim3 grid3((nx-1)/block3.x+1,ny);
    iStart=Seconds();
    sumMatrix<<<grid3,block3>>>(d_A,d_B,d_C,nx,ny);
    CHECK(hipDeviceSynchronize());
    iElaps=Seconds()-iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid3.x,grid3.y,block3.x,block3.y,iElaps);
    CHECK(hipMemcpy(h_C_from_gpu,d_C,nBytes,hipMemcpyDeviceToHost));
    checkResult(h_C,h_C_from_gpu,nxy);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_from_gpu);
    hipDeviceReset();
    return 0;    
}