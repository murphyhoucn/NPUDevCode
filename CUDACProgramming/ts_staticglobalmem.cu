#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ float dev_data;

__global__ void CheckGlobalVariable()
{
    printf("Device: the value of the global variable is %f\n", dev_data);
    dev_data += 2.0;
}

int main()
{
    float value = 3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(dev_data), &value, sizeof(float));
    printf("Host: copy %f to the global variable\n", value);

    CheckGlobalVariable<<<1, 1>>>();
    hipMemcpyFromSymbol(&value, HIP_SYMBOL(dev_data), sizeof(float));
    printf("Host: the value changed by the kernel to %f \n", value);
    hipDeviceReset();
    return EXIT_SUCCESS;
}