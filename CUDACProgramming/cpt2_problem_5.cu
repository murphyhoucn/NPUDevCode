#include "hip/hip_runtime.h"
// 习题 5
// Code 2-8
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // detect the cuda capabled devices
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n -> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (device_count == 0)
    {
        printf("There are no available devices that support CUDA!\n");
    }
    else
    {
        printf("Detected %d CUDA Capable devices.\n", device_count);
    }

    int dev, driver_version = 0, runtime_version = 0;

    dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, dev);
    printf("Device %d: %s\n", dev, device_prop.name);

    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);
    printf("CUDA Driver Version：%d.%d\n", driver_version / 1000, (driver_version % 100) / 10);
    printf("CUDA Runtime Version：%d.%d\n", runtime_version / 1000, (runtime_version % 100) / 10);

    printf("CUDA Capability Major/Minor version number: %d.%d\n", device_prop.major, device_prop.minor);

    printf("Total amount of global memory: %.2f MBytes (%llu bytes)\n", (float)device_prop.totalGlobalMem / (pow(1024.0, 0.3)), (unsigned long long)device_prop.totalGlobalMem);

    printf("GPU Clock rate: %.0f Mhz (%.2f Ghz)\n", device_prop.clockRate * 1e-3f, device_prop.clockRate * 1e-6f);
    printf("Memory Clock rate: %.0f Mhz\n", device_prop.memoryClockRate * 1e-3f);
    printf("Memory Bus Width: %d-bit\n", device_prop.memoryBusWidth);

    if (device_prop.l2CacheSize)
    {
        printf("L2 Cache Size %d bytes\n", device_prop.l2CacheSize);
    }

    printf("Max Texture Deimension Size (x, y, z): 1D = (%d), 2D = (%d, %d), 3D = (%d, %d, %d)\n", device_prop.maxTexture1D, device_prop.maxTexture2D[0], device_prop.maxTexture2D[1], device_prop.maxTexture3D[0], device_prop.maxTexture3D[1], device_prop.maxTexture3D[2]);
    printf("Max Layerd Texture Size (dim) x layers: 1D = (%d) x %d, 2D = (%d, %d) x %d\n", device_prop.maxTexture1DLayered[0], device_prop.maxTexture1DLayered[1], device_prop.maxTexture2DLayered[0], device_prop.maxTexture1DLayered[1], device_prop.maxTexture1DLayered[2]);

    printf("Total amount of constant memory: %lu bytes\n", device_prop.totalConstMem);
    printf("Total amount of shared memory per block: %lu bytes\n", device_prop.sharedMemPerBlock);
    printf("Total number of registers available per block: %d\n", device_prop.regsPerBlock);

    printf("Warp size: %d\n", device_prop.warpSize);
    printf("Maximum number of threads per multiprocessor: %d\n", device_prop.maxThreadsPerMultiProcessor);
    printf("Maximum number of threads per clock: %d\n", device_prop.maxThreadsPerBlock);
    printf("Maximum sizes of each dimension of a block: %d x %d x %d\n", device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1], device_prop.maxThreadsDim[2]);
    printf("Maximum sizes of each dimension of a grid: %d x %d x %d\n", device_prop.maxGridSize[0], device_prop.maxGridSize[1], device_prop.maxGridSize[2]);

    printf("Maximum memory pitch: %lu bytes\n", device_prop.memPitch);

    exit(EXIT_SUCCESS);
}

/*
Detected 8 CUDA Capable devices.
Device 0: NVIDIA GeForce RTX 3080
CUDA Driver Version：12.0
CUDA Runtime Version：11.3
CUDA Capability Major/Minor version number: 8.6
Total amount of global memory: 1311956992.00 MBytes (10495655936 bytes)
GPU Clock rate: 1710 Mhz (1.71 Ghz)
Memory Clock rate: 9501 Mhz
Memory Bus Width: 320-bit
L2 Cache Size 5242880 bytes
Max Texture Deimension Size (x, y, z): 1D = (131072), 2D = (131072, 65536), 3D = (16384, 16384, 16384)
Max Layerd Texture Size (dim) x layers: 1D = (32768) x 2048, 2D = (32768, 2048) x 32768
Total amount of constant memory: 65536 bytes
Total amount of shared memory per block: 49152 bytes
Total number of registers available per block: 65536
Warp size: 32
Maximum number of threads per multiprocessor: 1536
Maximum number of threads per clock: 1024
Maximum sizes of each dimension of a block: 1024 x 1024 x 64
Maximum sizes of each dimension of a grid: 2147483647 x 65535 x 65535
Maximum memory pitch: 2147483647 bytes
*/