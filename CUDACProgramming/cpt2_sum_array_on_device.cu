// Code 2-4

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

void SumArrayOnHost(float *array_a, float *array_b, float *array_c, const int kN)
{
    for (int idx = 0; idx < kN; ++idx)
    {
        array_c[idx] = array_a[idx] + array_b[idx];
    }
}

__global__ void SumArrayOnDevice(float *array_a, float *array_b, float *array_c, const int kN)
{
    for (int idx = 0; idx < kN; ++idx)
    {
        array_c[idx] = array_a[idx] + array_b[idx];
    }
}

void InitialArray(float *ip, int size)
{
    time_t t;
    srand((unsigned int)time(&t));

    for (int i = 0; i < size; ++i)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

int main()
{
    int n_element = 1024;
    size_t n_bytes = n_element * sizeof(float);

    float *h_array_a, *h_array_b, *h_array_c;
    h_array_a = (float *)malloc(n_bytes);
    h_array_b = (float *)malloc(n_bytes);
    h_array_c = (float *)malloc(n_bytes);

    float *gpu_ref;
    gpu_ref = (float *)malloc(n_bytes);

    float *d_array_a, *d_array_b, *d_array_c;
    hipMalloc((float **)&d_array_a, n_bytes);
    hipMalloc((float **)&d_array_b, n_bytes);
    hipMalloc((float **)&d_array_c, n_bytes);

    InitialArray(h_array_a, n_element);
    InitialArray(h_array_b, n_element);

    SumArrayOnHost(h_array_a, h_array_b, h_array_c, n_element);

    hipMemcpy(d_array_a, h_array_a, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_array_b, h_array_b, n_bytes, hipMemcpyHostToDevice);

    SumArrayOnDevice<<<1, 10>>>(d_array_a, d_array_b, d_array_c, n_element);

    hipMemcpy(gpu_ref, d_array_c, n_bytes, hipMemcpyDeviceToHost);

    free(h_array_a);
    free(h_array_b);
    free(h_array_c);

    hipFree(d_array_a);
    hipFree(d_array_b);
    hipFree(d_array_c);

    hipDeviceReset();

    return 0;
}