#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/common.h"

void SumArrays(float *a, float *b, float *c, const int size)
{
    for (int i = 0; i < size; i += 4)
    {
        c[i] = a[i] + b[i];
        c[i + 1] = a[i + 1] + b[i + 1];
        c[i + 2] = a[i + 2] + b[i + 2];
        c[i + 3] = a[i + 3] + b[i + 3];
    }
}

__global__ void SumArraysGPU(float *a, float *b, float *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = a[idx] + b[idx];
}

int main(int argc, char **argv)
{
    int dev = 0;
    hipSetDevice(dev);

    int power = 10;
    if (argc >= 2)
        power = atoi(argv[1]);
    int nElem = 1 << power;
    printf("Vector size:%d\n", nElem);
    int nByte = sizeof(float) * nElem;

    float *res_from_gpu_h = (float *)malloc(nByte);
    float *res_h = (float *)malloc(nByte);
    memset(res_h, 0, nByte);
    memset(res_from_gpu_h, 0, nByte);

    float *a_host, *b_host, *res_d;
    double iStart, iElaps;
    dim3 block(1024);
    dim3 grid(nElem / block.x);
    res_from_gpu_h = (float *)malloc(nByte);

    CHECK(hipMallocManaged((float **)&a_host, nByte, hipHostMallocMapped));
    CHECK(hipMallocManaged((float **)&b_host, nByte, hipHostMallocMapped));

    CHECK(hipMalloc((float **)&res_d, nByte));

    initialData(a_host, nElem);
    initialData(b_host, nElem);

    //=============================================================//
    iStart = cpuSecond();
    SumArraysGPU<<<grid, block>>>(a_host, b_host, res_d); // 统一内存寻址  Uniform Memory Access
    CHECK(hipMemcpy(res_from_gpu_h, res_d, nByte, hipMemcpyDeviceToHost));
    iElaps = cpuSecond() - iStart;
    //=============================================================//
    printf("UMA memory elapsed %lf ms \n", iElaps);
    printf("Execution configuration<<<%d,%d>>>\n", grid.x, block.x);

    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    //-----------------------normal memory---------------------------
    float *a_h_n = (float *)malloc(nByte);
    float *b_h_n = (float *)malloc(nByte);
    float *res_h_n = (float *)malloc(nByte);
    float *res_from_gpu_h_n = (float *)malloc(nByte);
    memset(res_h_n, 0, nByte);
    memset(res_from_gpu_h_n, 0, nByte);

    float *a_d_n, *b_d_n, *res_d_n;
    CHECK(hipMalloc((float **)&a_d_n, nByte));
    CHECK(hipMalloc((float **)&b_d_n, nByte));
    CHECK(hipMalloc((float **)&res_d_n, nByte));

    initialData(a_h_n, nElem);
    initialData(b_h_n, nElem);
    //=============================================================//
    iStart = cpuSecond();
    CHECK(hipMemcpy(a_d_n, a_h_n, nByte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d_n, b_h_n, nByte, hipMemcpyHostToDevice));
    SumArraysGPU<<<grid, block>>>(a_d_n, b_d_n, res_d_n);
    CHECK(hipMemcpy(res_from_gpu_h, res_d, nByte, hipMemcpyDeviceToHost));
    iElaps = cpuSecond() - iStart;
    //=============================================================//
    printf("device memory elapsed %lf ms \n", iElaps);
    printf("Execution configuration<<<%d,%d>>>\n", grid.x, block.x);
    //--------------------------------------------------------------------

    SumArrays(a_host, b_host, res_h, nElem);
    checkResult(res_h, res_from_gpu_h, nElem);

    hipHostFree(a_host);
    hipHostFree(b_host);
    hipFree(res_d);
    free(res_h);
    free(res_from_gpu_h);

    hipFree(a_d_n);
    hipFree(b_d_n);
    hipFree(res_d_n);

    free(a_h_n);
    free(b_h_n);
    free(res_h_n);
    free(res_from_gpu_h_n);
    return 0;
}