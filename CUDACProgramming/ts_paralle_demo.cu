#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

#include "common/common.h"

__global__ void SumMatrix(float *A, float *B, float *C, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x + blockDim.x;
    int iy = threadIdx.y + blockIdx.y + blockDim.y;
    int ixy = ix + iy * nx;

    if (ix < nx && iy < ny)
        C[ixy] = A[ixy] + B[ixy];
}

int main(int argc, char **argv)
{
    int devidx = 0;
    hipSetDevice(devidx);

    int nx = 1 << 13;
    int ny = 1 << 13;
    int nxy = nx * ny;
    int nbytes = nxy * sizeof(float);

    float *h_A = (float *)malloc(nbytes);
    float *h_B = (float *)malloc(nbytes);
    float *h_C = (float *)malloc(nbytes);
    float *h_C_gpu = (float *)malloc(nbytes);
    memset(h_C, 0, nbytes);
    memset(h_C_gpu, 0, nbytes);

    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;

    CHECK(hipMalloc((void **)&d_A, nbytes));
    CHECK(hipMalloc((void **)&d_B, nbytes));
    CHECK(hipMalloc((void **)&d_C, nbytes));

    CHECK(hipMemcpy(d_A, h_A, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nbytes, hipMemcpyHostToDevice));

    int dimx = argc > 2 ? atoi(argv[1]) : 32;
    int dimy = argc > 2 ? atoi(argv[2]) : 32;

    double t_start, t_elaps;

    dim3 block(dimx, dimy);
    dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);

    t_start = Seconds();
    SumMatrix<<<grid, block>>>(d_A, d_B, d_C, nx, ny);
    CHECK(hipDeviceSynchronize());
    t_elaps = Seconds() - t_start;

    printf("GPU Execution configuration <<<(%d, %d), (%d, %d)>>>  time: %f\n", grid.x, grid.y, block.x, block.y, t_elaps);

    CHECK(hipMemcpy(h_C_gpu, d_C, nbytes, hipMemcpyDeviceToHost));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_gpu);

    hipDeviceReset();

    return 0;
}