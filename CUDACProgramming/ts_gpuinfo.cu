#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n -> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FALL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    int dev = 0, driverVersion = 0, runtimeVersion = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d:\"%s\"\n", dev, deviceProp.name);

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("CUDA Driver Version / Runtime Version:       %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("CUDA Capability Major/Minor version number:  %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Total amount of global memory:               %.2f GBytes\n", (float)deviceProp.totalGlobalMem / pow(1024, 3));
    printf("GPU Clock rate:                              %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
    printf("Memory Bus width:                            %d-bits\n", deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize)
        printf("L2 Cache Size:                               %d bytes (%f Mbytes)\n", deviceProp.l2CacheSize, deviceProp.l2CacheSize / 1024.0 / 1024.0);

    printf("Max Texture Dimension Size (x,y,z)           1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n", deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
    printf("Max Layered Texture Size (dim) x layers      1D=(%d) x %d, 2D=(%d, %d) x %d \n", deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1], deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);

    printf("Total amount of constant memory              %lu bytes(%f kbytes)\n", deviceProp.totalConstMem, deviceProp.totalConstMem / 1024.0);                // 常量内存
    printf("Total amout of shared memory per block:      %ld bytes(%f kbytes)\n", deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerBlock / 1024.0);        // 共享内存
    printf("Total amout of Register per SM:              %d bytes(%f kbytes)\n", deviceProp.regsPerMultiprocessor, deviceProp.regsPerMultiprocessor / 1024.0); // 寄存器

    printf("Wrap size:                                   %d\n", deviceProp.warpSize);
    printf("Maximum number of thread per multiprocesser  %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of thread per block:          %d\n", deviceProp.maxThreadsPerBlock);
    printf("Maximum size of each dimension of a block:   %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("Maximum size of each dimension of a grid:    %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("Maximum memory pitch:                        %lu bytes\n", deviceProp.memPitch); // 最大连续线性内存

    printf("The number of multiprocessors(SM):           %d\n", deviceProp.multiProcessorCount);
    printf("Total amount of constant memeory:            %4.2f KB\n", deviceProp.totalConstMem / 1024.0);
    printf("Total amount of shared memeory per block:    %4.2f KB\n", deviceProp.sharedMemPerBlock / 1024.0);
    printf("Total num of registers available per block:  %f KB\n", deviceProp.regsPerBlock / 1024.0);

    printf("Bool: %d\n", deviceProp.localL1CacheSupported);
    printf("Bool: %d\n", deviceProp.globalL1CacheSupported);

    exit(EXIT_SUCCESS);
}

/*

GA 102 的手册
https://www.nvidia.com/content/PDF/nvidia-ampere-ga-102-gpu-architecture-whitepaper-v2.pdf

https://www.techpowerup.com/gpu-specs/geforce-rtx-3080.c3621
*/