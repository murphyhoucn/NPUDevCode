#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double CpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.0e-6);
}

__global__ void MathKernel1(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if (tid % 2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void MathKernel2(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void MathKernel3(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;
    bool pred = (tid % 2 == 0);
    if (pred)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void WarmUp(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[tid] = a + b;
}
int main(int argc, char **argv)
{
    int dev = 0;
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, dev);
    printf("%s using Device %d : %s\n", argv[0], dev, device_prop.name);

    int data_size = 64;
    int block_size = 64;
    printf("Data size : %d\n", data_size);

    dim3 block(block_size, 1);
    dim3 grid((data_size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d, grid %d)\n", block.x, grid.x);

    float *d_c;
    size_t n_bytes = data_size * sizeof(float);
    hipMalloc((float **)&d_c, n_bytes);

    double i_start, i_elaps;
    hipDeviceSynchronize();

    // warmup
    i_start = CpuSecond();
    WarmUp<<<grid, block>>>(d_c);
    hipDeviceSynchronize();
    i_elaps = CpuSecond() - i_start;
    printf("WarmUp      <<<%d, %d>>> elapsed %f sec.\n", grid.x, block.x, i_elaps);

    // MathKernel1
    i_start = CpuSecond();
    MathKernel1<<<grid, block>>>(d_c);
    hipDeviceSynchronize();
    i_elaps = CpuSecond() - i_start;
    printf("MathKernel1 <<<%d, %d>>> elapsed %f sec.\n", grid.x, block.x, i_elaps);

    // MathKernel2
    i_start = CpuSecond();
    MathKernel2<<<grid, block>>>(d_c);
    hipDeviceSynchronize();
    i_elaps = CpuSecond() - i_start;
    printf("MathKernel2 <<<%d, %d>>> elapsed %f sec.\n", grid.x, block.x, i_elaps);

    // MathKernel3
    i_start = CpuSecond();
    MathKernel3<<<grid, block>>>(d_c);
    hipDeviceSynchronize();
    i_elaps = CpuSecond() - i_start;
    printf("MathKernel3 <<<%d, %d>>> elapsed %f sec.\n", grid.x, block.x, i_elaps);

    hipFree(d_c);
    hipDeviceReset();
    return 0;
}