#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common/common.h"

/*
 * This code implements the interleaved and neighbour-paired approaches to parallel reduction in CUDA.
*/

// CPU Reduction
unsigned RecursiveReduce(int *data, int const kSize)
{
    // terminate check
    if (kSize == 1) return(data[0]);

    // renew the stride
    int const kStride = kSize / 2;
    
    // in-place reduction
    for (int i = 0; i < kStride; ++i)
    {
        data[i] += data[i + kStride];
    }

    // call recursicely
    return RecursiveReduce(data, kStride);
}

// Neighboured pair implementation with divergence
__global__ void ReduceNeighboured(int *g_idata, int *g_odata, unsigned int kN)
{
    unsigned int t_id = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // define idata as a pointer
    int *idata = g_idata + blockIdx.x * blockDim.x;

    if (idx >= kN) return;
    
    // 这段代码实现了分块内的并行归约操作。
    for(int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if((t_id % (2 * stride)) == 0)
        {
           idata[t_id] += idata[t_id + stride]; 
        }
        __syncthreads(); // 是一个同步原语，确保块内所有线程在继续下一步之前都完成当前步的计算。
    }

    if(t_id == 0) g_odata[blockIdx.x] = idata[0];
}

// Neighboured pair implementation with divergence
// WarmUp function is same to ReduceNeighboured
__global__ void WarmUp(int *g_idata, int *g_odata, unsigned int kN)
{
    unsigned int t_id = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // define idata as a pointer
    int *idata = g_idata + blockIdx.x * blockDim.x;

    if (idx >= kN) return;
    
    for(int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if((t_id % (2 * stride)) == 0)
        {
           idata[t_id] += idata[t_id + stride]; 
        }
        __syncthreads();
    }

    if(t_id == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char **argv)
{
    int dev = 0;
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d : %s ", dev, device_prop.name);
    hipSetDevice(dev);

    bool result_flag = false;

    // initialization
    int array_size = 1<<24;
    printf("with array size %d \n", array_size);

    // execution configuration
    int block_size = 512;
    if(argc > 1){block_size = atoi(argv[1]);}

    dim3 block (block_size, 1);
    dim3 grid ((array_size + block.x - 1) / block.x, 1);
    printf("grid : %d, block : %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = array_size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x * sizeof(int));
    int *tmp = (int *)malloc(bytes);

    // initialization the array
    for (int i = 0; i < array_size; ++i)
    {
        h_idata[i] = (int)(rand() & 0xFF); // 返回一个范围在 0 到 255 之间的整数
    }
    memcpy(tmp, h_idata, bytes);

    double i_start, i_elaps;
    unsigned gpu_sum = 0;
    
    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **)&d_idata, bytes));
    CHECK(hipMalloc((void **)&d_odata, bytes));

    // cpu reduction
    i_start = Seconds();
    unsigned cpu_sum = RecursiveReduce(tmp, array_size);
    i_elaps = Seconds() - i_start;
    printf("cpu reduce      elapsed %f ms. cpu_sum: %u\n", i_elaps * 1000, cpu_sum);

    // kernel 1: reduce neighboured(warm up)
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    i_start = Seconds();
    WarmUp<<<grid, block>>>(d_idata, d_odata, array_size);
    CHECK(hipDeviceSynchronize());
    i_elaps = Seconds() - i_start;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));

    gpu_sum = 0;
    for(int i = 0; i < grid.x; ++i) gpu_sum += h_odata[i];
    printf("gpu warmup      elapsed %f ms. gpu_sum = %u. <<<grid: %d, block:%d>>>. \n", i_elaps * 1000, gpu_sum, grid.x, block.x);

    // kernel 1: reduce neighboured
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    i_start = Seconds();
    ReduceNeighboured<<<grid, block>>>(d_idata, d_odata, array_size);
    CHECK(hipDeviceSynchronize());
    i_elaps = Seconds() - i_start;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    
    gpu_sum = 0;
    for(int i = 0; i < grid.x; ++i) gpu_sum += h_odata[i];
    printf("gpu Reduce      elapsed %f ms. gpu_sum = %u. <<<grid: %d, block:%d>>>. \n", i_elaps * 1000, gpu_sum, grid.x, block.x);

    CHECK(hipDeviceSynchronize());
    i_elaps = Seconds() - i_start;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for(int i = 0; i < grid.x / 8; ++i) gpu_sum += h_odata[i];
    printf("gpu Cmptnroll   elapsed %f ms. gpu_sum = %u. <<<grid: %d, block:%d>>>. \n", i_elaps * 1000, gpu_sum, grid.x, block.x);

    // free host memeory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // rest device
    CHECK(hipDeviceReset());

    // check the result
    result_flag = (gpu_sum == cpu_sum);
    if(!result_flag) printf("Test failed!\n");

    return EXIT_SUCCESS;
}