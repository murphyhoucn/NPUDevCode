#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "common/common.h"

__global__ void WarmUp(float* C)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float a = 0.0f, b = 0.0f;

    if((tid/warpSize) % 2 == 0) a = 100.0f;
    else b = 200.0f;

    C[tid] = a + b;
}

__global__ void MathKernek1(float* C)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float a = 0.0f, b = 0.0f;

    if(tid % 2 == 0) a = 100.0f;
    else b = 200.0f;

    C[tid] = a + b;
}
__global__ void MathKernek2(float* C)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float a = 0.0f, b = 0.0f;

    if((tid/warpSize) % 2 == 0) a = 100.0f;
    else b = 200.0f;

    C[tid] = a + b;
}
__global__ void MathKernek3(float* C)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float a = 0.0f, b = 0.0f;
    bool ipred = (tid % 2) == 0;

    if(ipred) a = 100.0f;
    else b = 200.0f;

    C[tid] = a + b;
}


int main(int argc, char** argv)
{
    int dev = 0;
    hipDeviceProp_t deviceprop;
    hipGetDeviceProperties(&deviceprop, dev);
    printf("Using Device %d:%s", dev, deviceprop.name);

    int datasize = 64;
    int blocksize = 64;

    if(argc > 1) datasize = atoi(argv[1]);
    if(argc > 2) blocksize = atoi(argv[2]);

    dim3 block(blocksize, 1);
    dim3 grid((datasize - 1) / block.x + 1, 1);
    printf("Execuation configuration(block, grid): (%d, %d)\n", block.x, grid.x);

    float* d_C;
    size_t nbytes = datasize * sizeof(float);
    float* h_C = (float*)malloc(nbytes);
    hipMalloc((float**)&d_C, nbytes);

    // warmup kernel
    double t_start, t_elaps;
    hipDeviceSynchronize();
    t_start = Seconds();
    WarmUp<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    t_elaps = Seconds() - t_start;
    printf("WarmUp      <<<%d,%d>>> elaps %lf sec \n", grid.x, block.x, t_elaps);

    // 1
    hipDeviceSynchronize();
    t_start = Seconds();
    MathKernek1<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    t_elaps = Seconds() - t_start;
    printf("MathKernek1 <<<%d,%d>>> elaps %lf sec \n", grid.x, block.x, t_elaps);

    // 2
    hipDeviceSynchronize();
    t_start = Seconds();
    MathKernek2<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    t_elaps = Seconds() - t_start;
    printf("MathKernek2 <<<%d,%d>>> elaps %lf sec \n", grid.x, block.x, t_elaps);

    // 3
    hipDeviceSynchronize();
    t_start = Seconds();
    MathKernek3<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    t_elaps = Seconds() - t_start;
    printf("MathKernek3 <<<%d,%d>>> elaps %lf sec \n", grid.x, block.x, t_elaps);    
    
    free(h_C);
    hipFree(d_C);
    hipDeviceReset();
    
    return 0;
}