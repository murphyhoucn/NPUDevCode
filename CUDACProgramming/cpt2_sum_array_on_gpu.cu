#include "hip/hip_runtime.h"
// Code 2-4
#include <hip/hip_runtime.h>
#include <stdio.h>

// '/'的意思是换行符，一行代码写成了多行
#define CHECK(call)                                                             \
    {                                                                           \
        const hipError_t error = call;                                         \
        if (error != hipSuccess)                                               \
        {                                                                       \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
            printf("cude: %d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                            \
        }                                                                       \
    }

void CheckResult(float *host_ref, float *gpu_ref, const int kN)
{
    double epsilon = 1.0e-8;
    bool match = 1;

    for (int i = 0; i < kN; ++i)
    {
        if (abs(host_ref[i] - gpu_ref[i]) > epsilon)
        {
            match = 0;
            printf("Array do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", host_ref[i], gpu_ref[i], i);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
}

void InitData(float *ip, int size)
{
    // generate different seed for random number.
    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; ++i)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void SumArrayOnHost(float *a, float *b, float *c, const int kN)
{
    for (int idx = 0; idx < kN; ++idx)
    {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void SumArrayOnDevice(float *a, float *b, float *c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    // set device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vector
    int n_element = 32;
    printf("Vector size %d\n", n_element);

    // malloc host memory
    size_t n_bytes = n_element * sizeof(float);

    float *h_a, *h_b, *host_ref, *gpu_ref;
    h_a = (float *)malloc(n_bytes);
    h_b = (float *)malloc(n_bytes);
    host_ref = (float *)malloc(n_bytes);
    gpu_ref = (float *)malloc(n_bytes);

    // initialize data at host side
    InitData(h_a, n_element);
    InitData(h_b, n_element);

    memset(host_ref, 0, n_bytes);
    memset(gpu_ref, 0, n_bytes);

    // malloc device global memoty
    float *d_a, *d_b, *d_c;
    hipMalloc((float **)&d_a, n_bytes);
    hipMalloc((float **)&d_b, n_bytes);
    hipMalloc((float **)&d_c, n_bytes);

    // transfer data from host to device
    hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    dim3 block(n_element);
    dim3 grid(n_element / block.x);

    SumArrayOnDevice<<<grid, block>>>(d_a, d_b, d_c);
    printf("Execution configuration<<<%d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    hipMemcpy(gpu_ref, d_c, n_bytes, hipMemcpyDeviceToHost);

    // add vector at host side for result checks
    SumArrayOnHost(h_a, h_b, host_ref, n_element);

    // check device results
    CheckResult(host_ref, gpu_ref, n_element);

    // free device global memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // free host memory
    free(h_a);
    free(h_b);
    free(host_ref);
    free(gpu_ref);

    return 0;
}