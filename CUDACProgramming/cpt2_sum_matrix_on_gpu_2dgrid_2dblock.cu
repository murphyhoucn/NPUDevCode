#include "hip/hip_runtime.h"
// Code 2-7
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call)                                                                                               \
    {                                                                                                             \
        const hipError_t error = call;                                                                           \
        if (error != hipSuccess)                                                                                 \
        {                                                                                                         \
            printf("Error: %s:%d, code: %d, reason: %s\n", __FILE__, __LINE__, error, hipGetErrorString(error)); \
            exit(-10 * error);                                                                                    \
        }                                                                                                         \
    }

void CheckResult(float *host_ref, float *gpu_ref, const int kN)
{
    double epsilon = 1.0e-8;
    bool match = 1;

    for (int i = 0; i < kN; ++i)
    {
        if (abs(host_ref[i] - gpu_ref[i]) > epsilon)
        {
            match = 0;
            printf("Matrix do no match!");
            printf("host: %5.2f gpu: %5.2f at current %d\n", host_ref[i], gpu_ref[i], i);
            break;
        }
    }
    if (match)
        printf("Matrix math!\n");
}

double CpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.0e-6);
}

void InitData(float *ip, int size)
{
    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; ++i)
    {
        *(ip + i) = (float)(rand() & 0xFF) / 10.0f;
    }
}

void SumMatrixOnHost(float *a, float *b, float *c, const int knx, const int kny)
{
    for (int i = 0; i < knx; ++i)
    {
        for (int j = 0; j < kny; ++j)
        {
            *(c + i * kny + j) = *(b + i * kny + j) + *(a + i * kny + j);
        }
    }
}

__global__ void SumMatrixOnGPU2D(float *a, float *b, float *c, int knx, int kny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * knx + ix;

    if (ix < knx && iy < kny)
        *(c + idx) = *(a + idx) + *(b + idx);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t device_prop;
    CHECK(hipGetDeviceProperties(&device_prop, dev));
    printf("Using device %d : %s \n", dev, device_prop.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int n_bytes = nxy * sizeof(float);
    printf("Matrix size : (nx, ny): (%d, %d)\n", nx, ny);

    // malloc host memory
    float *h_a, *h_b, *host_ref, *gpu_ref;
    h_a = (float *)malloc(n_bytes);
    h_b = (float *)malloc(n_bytes);
    host_ref = (float *)malloc(n_bytes);
    gpu_ref = (float *)malloc(n_bytes);

    // initialize data at host side
    double i_start = CpuSecond();
    InitData(h_a, nxy);
    InitData(h_b, nxy);
    double i_elaps = CpuSecond() - i_start;

    memset(host_ref, 0, n_bytes);
    memset(gpu_ref, 0, n_bytes);

    // add matrix at host side for result checks
    i_start = CpuSecond();
    SumMatrixOnHost(h_a, h_b, host_ref, nx, ny);
    i_elaps = CpuSecond() - i_start;

    // malloc device global memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, n_bytes);
    hipMalloc((void **)&d_b, n_bytes);
    hipMalloc((void **)&d_c, n_bytes);

    // transfer data from host to device
    hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    i_start = CpuSecond();
    SumMatrixOnGPU2D<<<grid, block>>>(d_a, d_b, d_c, nx, ny);
    hipDeviceSynchronize();
    i_elaps = CpuSecond() - i_start;
    printf("SumMatrixOnGPU2D<<<(%d, %d), (%d, %d)>>> elapsd %f sec.\n", grid.x, grid.y, block.x, block.y, i_elaps);

    // copy kernel result back to host
    hipMemcpy(gpu_ref, d_c, n_bytes, hipMemcpyDeviceToHost);

    // check device results
    CheckResult(host_ref, gpu_ref, nxy);

    // free device global memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // free host memory
    free(h_a);
    free(h_b);
    free(host_ref);
    free(gpu_ref);

    // reset device
    hipDeviceReset();

    return 0;
}