// Code 2-3
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int n_element = 1024;

    dim3 block(1024);
    dim3 grid = ((n_element + block.x - 1) / block.x);
    printf("grid.x = %d, block.x = %d\n", grid.x, block.x);

    block.x = 512;
    grid.x = (n_element + block.x - 1) / block.x;
    printf("grid.x = %d, block.x = %d\n", grid.x, block.x);

    block.x = 256;
    grid.x = (n_element + block.x - 1) / block.x;
    printf("grid.x = %d, block.x = %d\n", grid.x, block.x);

    block.x = 128;
    grid.x = (n_element + block.x - 1) / block.x;
    printf("grid.x = %d, block.x = %d\n", grid.x, block.x);

    hipDeviceReset();
    return 0;
}