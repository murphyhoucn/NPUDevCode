#include "hip/hip_runtime.h"
# include <stdio.h>
# include <hip/hip_runtime.h>
# include <stdlib.h>

#include "common/common.h"
__global__ void printThreadIdx(float* A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    unsigned int idx = ix + nx * iy;
    printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d) global index %2d ival %2f\n",threadIdx.x,threadIdx.y, blockIdx.x,blockIdx.y, ix,iy, idx, A[idx]);

}

int main()
{
    int dev_idx = 0;
    hipSetDevice(dev_idx);

    srand((unsigned int)time(NULL));

    int nx = 8, ny=6;
    int nxy = nx * ny;
    size_t nBytes = nxy * sizeof(int);

    float* h_A = (float*)malloc(nBytes);

    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = (float)rand() / RAND_MAX;
    }

    // 打印二维数组
    for (int i = 0; i < nxy; i++) 
    {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    // device
    float* d_A = NULL;
    CHECK(hipMalloc((void**)&d_A, nBytes));
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);

    printThreadIdx<<<grid, block>>>(d_A, nx, ny);

    CHECK(hipDeviceSynchronize());
    hipFree(d_A);
    free(h_A);

    hipDeviceReset();

    return 0;
}