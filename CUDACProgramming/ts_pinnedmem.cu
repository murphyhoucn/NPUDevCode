#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/common.h"

void SumArrays(float *a, float *b, float *c, const int size)
{
    for (int i = 0; i < size; i += 4)
    {
        c[i] = a[i] + b[i];
        c[i + 1] = a[i + 1] + b[i + 1];
        c[i + 2] = a[i + 2] + b[i + 2];
        c[i + 3] = a[i + 3] + b[i + 3];
    }
}

__global__ void SumArraysGPU(float *a, float *b, float *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = a[idx] + b[idx];
}

int main()
{
    int devidx = 0;
    hipSetDevice(devidx);

    int n_element = 1 << 14;
    printf("Vector size = %d\n", n_element);

    size_t n_bytes = sizeof(int) * n_element;

    float *h_a = (float *)malloc(n_bytes);
    float *h_b = (float *)malloc(n_bytes);
    float *h_c = (float *)malloc(n_bytes);
    float *h_c_gpu = (float *)malloc(n_bytes);

    memset(h_c, 0, n_bytes);
    memset(h_c_gpu, 0, n_bytes);
    initialData(h_a, n_element);
    initialData(h_b, n_element);

    // float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    // CHECK(hipHostMalloc((float **)&d_a, n_bytes));
    // CHECK(hipHostMalloc((float **)&d_b, n_bytes));
    // CHECK(hipHostMalloc((float **)&d_c, n_bytes));

    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    CHECK(hipMalloc((float **)&d_a, n_bytes));
    CHECK(hipMalloc((float **)&d_b, n_bytes));
    CHECK(hipMalloc((float **)&d_c, n_bytes));

    CHECK(hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice));

    dim3 block(1024);
    dim3 grid((n_element - block.x) / block.x + 1);

    SumArraysGPU<<<grid, block>>>(d_a, d_b, d_c);
    printf("Execution configuration<<<%d,%d>>>\n", grid.x, block.x);
    CHECK(hipMemcpy(h_c_gpu, d_c, n_bytes, hipMemcpyDeviceToHost));
    SumArrays(h_a, h_b, h_c, n_element);

    checkResult(h_c, h_c_gpu, n_element);
    hipHostFree(d_a);
    hipHostFree(d_b);
    hipHostFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_gpu);

    return 0;
}