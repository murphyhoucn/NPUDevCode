#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "common/common.h"

void InitialData(float *ip, const int size)
{
    int i;
    for (i = 0; i < size; i++)
    {
        *(ip + i) = (float)(rand() & 0xFF) / 10.0f;
    }
}

void SumMatrixOnHost(float *a, float *b, float *c, const int knx, const int kny)
{
    for (int iy = 0; iy < kny; ++iy)
    {
        for (int ix = 0; ix < knx; ++ix)
        {
            *(c + ix) = *(a + ix) + *(b + ix);
        }
        c += knx;
        b += knx;
        a += knx;
    }
}

void CheckResult(float *host_ref, float *device_ref, const int kn)
{
    double epsilon = 1.0e-8;
    for (int i = 0; i < kn; ++i)
    {
        if (abs(*(host_ref + i) - *(device_ref + i)) > epsilon)
        {
            printf("host: %f; device: %f\n", *(host_ref + i), *(device_ref + i));
            printf("Arrays do not match!\n");
            break;
        }
    }
}

__global__ void SumMatrixOnGPU2D(float *a, float *b, float *c, const int knx, const int kny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * knx + ix;

    if (ix < knx && ix < kny)
    {
        *(c + idx) = *(b + idx) + *(a + idx);
    }
}

int main(int argc, char **argv)
{
    // set device
    int dev = 0;
    hipDeviceProp_t device_prop;
    CHECK(hipGetDeviceProperties(&device_prop, dev));
    CHECK(hipSetDevice(dev));
    printf("%s using Device %d : %s\n", argv[0], dev, device_prop.name);

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;
    int nxy = nx * ny;
    int n_bytes = nxy * sizeof(float);

    // malloc host memory
    float *h_a, *h_b, *host_ref, *device_ref;
    h_a = (float *)malloc(n_bytes);
    h_b = (float *)malloc(n_bytes);
    host_ref = (float *)malloc(n_bytes);
    device_ref = (float *)malloc(n_bytes);

    // initialize data at host side
    size_t i_start = Seconds();
    InitialData(h_a, nxy);
    InitialData(h_b, nxy);
    size_t i_elaps = Seconds() - i_start;

    memset(host_ref, 0, n_bytes);
    memset(device_ref, 0, n_bytes);

    // add matrix at host side for result checks
    i_start = Seconds();
    SumMatrixOnHost(h_a, h_b, host_ref, nx, ny);
    i_elaps = Seconds() - i_start;

    // malloc device global memory
    float *d_a, *d_b, *d_c;
    CHECK(hipMalloc((void **)&d_a, n_bytes));
    CHECK(hipMalloc((void **)&d_b, n_bytes));
    CHECK(hipMalloc((void **)&d_c, n_bytes));

    // transfer datat from host to device
    CHECK(hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;

    if (argc > 2)
    {
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // excute the kernal
    CHECK(hipDeviceSynchronize());
    i_start = Seconds();
    SumMatrixOnGPU2D<<<block, grid>>>(d_a, d_b, d_c, nx, ny);
    CHECK(hipDeviceSynchronize());
    i_elaps = Seconds() - i_start;
    printf("SumMatrixOnGPU2D<<<(%d, %d), (%d, %d)>>> elapsed %f ms\n", grid.x, grid.y, block.x, block.y, (double)i_elaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(device_ref, d_c, n_bytes, hipMemcpyDeviceToHost));

    // check results
    CheckResult(host_ref, device_ref, nxy);

    // free device global memory
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));

    // free host memory
    free(h_a);
    free(h_b);
    free(host_ref);
    free(device_ref);

    // reset device
    CHECK(hipDeviceReset());

    return 0;
}