# include <stdio.h>
# include <hip/hip_runtime.h>

# define CHECK(call) \
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        printf("ERROR: %s:%d,", __FILE__, __LINE__);\
        printf("code:%d, resason:%s\n", error, hipGetErrorString(error));\
        exit(1);\
    }\
}\


void sumArrayOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; i++)
        C[i] = A[i] + B[i];
}

__global__ void sumArrayOnDevice(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const int N = 1024;
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *gpu_C = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    memset(h_C, 0, size);
    memset(gpu_C, 0, size);

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void **)&d_A, size));
    CHECK(hipMalloc((void **)&d_B, size));
    CHECK(hipMalloc((void **)&d_C, size));

    CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    
    sumArrayOnDevice<<<1,N>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(gpu_C, d_C, size, hipMemcpyDeviceToHost));

    sumArrayOnHost(h_A, h_B, h_C, N);

    for (int i = 0; i < N; i++) {
        if (gpu_C[i] != h_C[i]) {
            printf("Error: gpu_C[%d] = %f != %f = h_C[%d]\n", i, gpu_C[i], h_C[i], i);
            exit(EXIT_FAILURE);
        }
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(gpu_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
        
}
