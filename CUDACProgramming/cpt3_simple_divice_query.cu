#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int dev = 0;
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, dev);

    printf("Device %d : %s\n", dev, dev_prop.name);                                                     // GPU index and GPU Name
    printf("Number of multiprocessors: %d\n", dev_prop.multiProcessorCount);                            // SM
    printf("Total amount of constant memory: %4.2f KB\n", dev_prop.totalConstMem / 1024.0);             // Constant memory
    printf("Total amount of shared memory per block: %4.2f KB\n", dev_prop.sharedMemPerBlock / 1024.0); // Shared memory
    printf("Total number of registers available per block: %d\n", dev_prop.regsPerBlock);               // registers per block
    printf("Warp size: %d\n", dev_prop.warpSize);                                                       // warp size
    printf("Maximum number of threads per block: %d\n", dev_prop.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocessor: %d\n", dev_prop.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multiprocessor: %d\n", dev_prop.maxThreadsPerMultiProcessor / 32);

    return EXIT_SUCCESS;
}